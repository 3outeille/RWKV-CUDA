#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const _w, const F *__restrict__ const _u,
                               F *__restrict__ const _y)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _h = (idx / N) % H;
    const int _i = idx % N;

    const int _o0 = _b*T*C + _h*N;
    const int _o1 = _h*N;
    const F *__restrict__ const k = _k + _o0;
    const F *__restrict__ const v = _v + _o0 + _i;
    const F *__restrict__ const r = _r + _o0;
    F *__restrict__ const y = _y + _o0 + _i;

    float state[N] = {0};   

    for (int __t = 0; __t < T; __t++)
    {
        const int _t = __t*C;
        const F vv = v[_t];

        for (int _j = 0; _j < N; _j++) 
        {
            const int j = _t + _j;
            const int m = _o1 + _j;

            const float x = k[j] * vv;
            const float s = state[_j];
            
            atomicAdd(y + _t, r[j] * (_u[m] * x + s));
            state[_j] = s * _w[m] + x;
        }
    }
}

template <typename F>
__global__ void kernel_backward (const int B, const int T, const int C, const int H,
    const F *__restrict__ const r, const F *__restrict__ const k, const F *__restrict__ const v, const F *__restrict__ w, const F *__restrict__ const wwww, const F *__restrict__ u, const F *__restrict__ const gy,
    F *__restrict__ const gr, F *__restrict__ const gk, F *__restrict__ const gv, F *__restrict__ const gw, F *__restrict__ gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    w += h*N;
    u += h*N;
    gu += h*N;

    __shared__ float state[N * N], vv[N], gyy[N];

    for (int j = 0; j < N; ++j)
        state[j * N + i] = 0;
    
    const float ww = w[i];
    const float uu = u[i];

    for (int _t = b*T*C + h*N + i, _tend = (b+1)*T*C + h*N + i; _t < _tend; _t += C)
    {
        const F kk = k[_t];
        const F rr = r[_t];
        F grr = 0;
        F guu = 0;

        vv[i] = v[_t];
        gyy[i] = gy[_t];

        __syncthreads();

        for (int j = 0; j < N; j++)
        {

            float x = vv[j] * kk;
            float s = state[j * N + i];

            grr += gyy[j] * (uu * x + s);
            state[j * N + i] = s * ww + x;
            guu += rr * x * gyy[j];
        }
        gr[_t] = grr;
        atomicAdd(gu + i, guu);

        __syncthreads();
    }
}

void cuda_forward(int B, int T, int C, int H, float *r, float *k, float *v, float *w, float *u, float *y)
{
    dim3 threadsPerBlock( min(B*C, 32) );
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, float *r, float *k, float *v, float *w, float *ww, float *u, float *gy, float *gr, float *gk, float *gv, float *gw, float *gu)
{
    assert(H*N == C);
    kernel_backward<<<dim3(B * H), dim3(N)>>>(B, T, C, H, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}
