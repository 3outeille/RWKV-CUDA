#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                                      const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const F *__restrict__ const _w, const F *__restrict__ const _u,
                                      F *__restrict__ const _y)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _h = (idx / N) % H;
    const int _i = idx % N;

    const int _o0 = _b * T * C + _h * N;
    const int _o1 = _h * N;

    const float4 *__restrict__ const k = (float4 *)(_k + _o0);
    const float4 *__restrict__ const r = (float4 *)(_r + _o0);
    const float4 *__restrict__ const w = (float4 *)(_w + _o1);
    const float4 *__restrict__ const u = (float4 *)(_u + _o1);
    const F *__restrict__ const v = _v + _o0 + _i;
    F *__restrict__ const y = _y + _o0 + _i;

    __align__(16) float4 state[N / 4] = { make_float4(0.0f, 0.0f, 0.0f, 0.0f) };

    for (int __t = 0; __t < T; __t++)
    {
        const int _t = __t * (C >> 2);
        const int tt = __t * C;
        const F vv = v[tt];

        for (int _j = 0; _j < N / 4; _j++)
        {
            const int j = _t + _j;

            const float4 k_val = k[j];
            const float4 r_val = r[j];
            float4 x;
            x.x = k_val.x * vv;
            x.y = k_val.y * vv;
            x.z = k_val.z * vv;
            x.w = k_val.w * vv;

            float4 s = state[_j];

            float4 result;
            result.x = r_val.x * (u[_j].x * x.x + s.x);
            result.y = r_val.y * (u[_j].y * x.y + s.y);
            result.z = r_val.z * (u[_j].z * x.z + s.z);
            result.w = r_val.w * (u[_j].w * x.w + s.w);

            atomicAdd(&(y[tt]), result.x);
            atomicAdd(&(y[tt]), result.y);
            atomicAdd(&(y[tt]), result.z);
            atomicAdd(&(y[tt]), result.w);

            state[_j].x = s.x * w[_j].x + x.x;
            state[_j].y = s.y * w[_j].y + x.y;
            state[_j].z = s.z * w[_j].z + x.z;
            state[_j].w = s.w * w[_j].w + x.w;
        }
    }
}

template <typename F>
__global__ void kernel_backward (const int B, const int T, const int C, const int H,
    const F *__restrict__ const r, const F *__restrict__ const k, const F *__restrict__ const v, const F *__restrict__ const w, const F *__restrict__ const wwww, const F *__restrict__ const _u, const F *__restrict__ const gy,
    F *__restrict__ const gr, F *__restrict__ const gk, F *__restrict__ const gv, F *__restrict__ const gw, F *__restrict__ const gu)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // B * H * T * N
    const int b = idx / H / T / N;
    const int h = (idx / T / N) % H;
    const int t = (idx / N) % T;
    const int n = idx % N;
    
    for(int nn = 0; nn < N; nn++){
        for (int tt = 0; tt <= t; tt++) {
            F ww = (tt == t) ? _u[h*N + n] : pow(w[h*N + n], t-tt-1);
            
            gr[b*T*H*N + t*H*N + h*N + n] += ww * k[b*T*H*N + tt*H*N + h*N + n] *
                v[b*T*H*N + tt*H*N + h*N + nn] * gy[b*T*H*N + t*H*N + h*N + nn];
        }

        for (int tt = t; tt < T; tt++) {
            F ww = (tt == t) ? _u[h*N + n] : pow(w[h*N + n], tt-t-1);
            
            gk[b*T*H*N + t*H*N + h*N + n] += r[b*T*H*N + tt*H*N + h*N + n] * ww *
                v[b*T*H*N + t*H*N + h*N + nn] * gy[b*T*H*N + tt*H*N + h*N + nn];

            ww = (tt == t) ? _u[h*N + nn] : pow(w[h*N + nn], tt-t-1);
            
            gv[b*T*H*N + t*H*N + h*N + n] += r[b*T*H*N + tt*H*N + h*N + nn] * ww *
                k[b*T*H*N + t*H*N + h*N + nn] * gy[b*T*H*N + tt*H*N + h*N + n];
        }

        atomicAdd(gu + h*N + n, r[b*T*H*N + t*H*N + h*N + n] * k[b*T*H*N + t*H*N + h*N + n] *
                v[b*T*H*N + t*H*N + h*N + nn] * gy[b*T*H*N + t*H*N + h*N + nn]);

        for (int tt = 0; tt < t-1; tt++) {
            F ww = (t-tt-1) * wwww[h*N + n] * pow(w[h*N + n], t-tt-1);

            atomicAdd(gw + h*N + n, r[b*T*H*N + t*H*N + h*N + n] * ww * k[b*T*H*N + tt*H*N + h*N + n] *
                v[b*T*H*N + tt*H*N + h*N + nn] * gy[b*T*H*N + t*H*N + h*N + nn]);
        }
    }
}

void cuda_forward(int B, int T, int C, int H, float *r, float *k, float *v, float *w, float *u, float *y)
{
    assert(H*N == C);
    const int SIZE = B*C;
    dim3 threadsPerBlock(min(SIZE, 32));
    assert(SIZE % threadsPerBlock.x == 0);
    dim3 numBlocks(SIZE / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, float *r, float *k, float *v, float *w, float *ww, float *u, float *gy, float *gr, float *gk, float *gv, float *gw, float *gu)
{
    dim3 threadsPerBlock( min(B*H*T*N, 256) );
    assert(B * H * T * N % threadsPerBlock.x == 0);
    dim3 numBlocks(B * H * T * N / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, H, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}
