#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C, const int H,
                               const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const F *__restrict__ _u,
                               F *__restrict__ const _y)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;

    __shared__ float r[_N_], k[_N_], u[_N_], w[_N_];
    float state[_N_] = {0};

    __syncthreads();
    u[i] = float(_u[i]);
    w[i] = float(_w[i]);
    __syncthreads();

    for (int t = b*T*C + h*_N_ + i; t < (b+1)*T*C + h*_N_ + i; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        k[i] = float(_k[t]);
        __syncthreads();

        const float v = float(_v[t]);
        float y = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j+=4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& k_ = (float4&)(k[j]);
            const float4& w_ = (float4&)(w[j]);
            const float4& u_ = (float4&)(u[j]);
            float4& s = (float4&)(state[j]);
            float4 x;

            x.x = k_.x * v;
            x.y = k_.y * v;
            x.z = k_.z * v;
            x.w = k_.w * v;

            y += r_.x * (u_.x * x.x + s.x);
            y += r_.y * (u_.y * x.y + s.y);
            y += r_.z * (u_.z * x.z + s.z);
            y += r_.w * (u_.w * x.w + s.w);

            s.x = s.x * w_.x + x.x;
            s.y = s.y * w_.y + x.y;
            s.z = s.z * w_.z + x.z;
            s.w = s.w * w_.w + x.w;
        }
        _y[t] = F(y);
    }
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C, const int H,
    const F *__restrict__ const _r, const F *__restrict__ const _k, const F *__restrict__ const _v, const float *__restrict__ _w, const float *__restrict__ __w, const F *__restrict__ _u, const F *__restrict__ const _gy,
    F *__restrict__ const _gr, F *__restrict__ const _gk, F *__restrict__ const _gv, F *__restrict__ const _gw, F *__restrict__ const _gu)
{
    const int b = blockIdx.x / H;
    const int h = blockIdx.x % H;
    const int i = threadIdx.x;
    _w += h*_N_;
    _u += h*_N_;
    __w += h*_N_;
    const float w = _w[i];
    const float u = float(_u[i]);
    const float ww = __w[i];

    __shared__ float v[_N_], r[_N_], k[_N_], gy[_N_], gy2[_N_], w_[_N_], u_[_N_];    
    float state[_N_*2] = {0};

    float gw = 0, gu = 0;
    const int t000 = b*T*C + h*_N_ + i;
    const int t111 = (b+1)*T*C + h*_N_ + i;
    const int t222 = t111 - 2*C;

    for (int _t = t000; _t < t111; _t += C)
    {
        __syncthreads();
        v[i] = float(_v[_t]);
        gy[i] = float(_gy[_t]);
        __syncthreads();

        const float k = float(_k[_t]);
        const float r = float(_r[_t]);
        
        float gr = 0;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float x = v[j] * k;
            float s = state[j];
            state[j] = s * w + x;

            gr += gy[j] * (u * x + s);
            gu += r * x * gy[j];
        }
        _gr[_t] = F(gr);
    }
    _gu[b*C + h*_N_ + i] = F(gu);

    #pragma unroll
    for (int j = 0; j < _N_*2; ++j) {
        state[j] = 0;
    }
    
    for (int _t = t000; _t < t222; _t += C)
    {
        __syncthreads();
        v[i] = float(_v[_t]);
        gy2[i] = float(_gy[_t + 2*C]);
        __syncthreads();
        const float r2 = float(_r[_t + 2*C]);
        const float k = float(_k[_t]);

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            float x = v[j] * k;
            // accum[j] = w[h,i] * (accum[j] + accum[j+N] + x)
            // accum[j+N] = w[h,i] * (accum[j+N] + x)
            // gw[h,i] += r[b,t+2,h,i] * _w[h,i] * accum[j] * gy[b,t+2,h,j]
            state[j] = w * (state[j] + state[j+_N_] + x);
            state[j+_N_] = w * (state[j+_N_] + x);
            gw += r2 * ww * state[j] * gy2[j];
        }
    }
    
    _gw[b*C + h*_N_ + i] = F(gw);

    #pragma unroll
    for (int j = 0; j < _N_; ++j) {
        state[j] = 0;
    }

    __syncthreads();
    w_[i] = float(_w[i]);
    u_[i] = float(_u[i]);
    __syncthreads();
    
    for (int _t = t111 - C; _t >= t000; _t -= C)
    {
        __syncthreads();
        v[i] = float(_v[_t]);
        gy[i] = float(_gy[_t]);
        k[i] = float(_k[_t]);
        r[i] = float(_r[_t]);
        __syncthreads();

        float gk = 0, x, s;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            x = gy[j] * r[i];
            s = state[j];
            state[j] = s * w + x;
            gk += v[j] * (u * x + s);
        }
        _gk[_t] = F(gk);
    }

    #pragma unroll
    for (int j = 0; j < _N_; ++j) {
        state[j] = 0;
    }

    for (int _t = t111 - C; _t >= t000; _t -= C)
    {
        __syncthreads();
        gy[i] = float(_gy[_t]);
        r[i] = float(_r[_t]);
        k[i] = float(_k[_t]);
        __syncthreads();

        float gv = 0, x, s;

        #pragma unroll
        for (int j = 0; j < _N_; j++)
        {
            x = gy[i] * r[j];
            s = state[j];
            state[j] = s * w_[j] + x;
            gv += k[j] * (u_[j] * x + s);
        }
        _gv[_t] = F(gv);
    }
}

void cuda_forward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, bf16 *u, bf16 *y)
{
    assert(H*_N_ == C);
    kernel_forward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, u, y);
}

void cuda_backward(int B, int T, int C, int H, bf16 *r, bf16 *k, bf16 *v, float *w, float *ww, bf16 *u, bf16 *gy, bf16 *gr, bf16 *gk, bf16 *gv, bf16 *gw, bf16 *gu)
{
    assert(H*_N_ == C);
    kernel_backward<<<dim3(B * H), dim3(_N_)>>>(B, T, C, H, r, k, v, w, ww, u, gy, gr, gk, gv, gw, gu);
}
